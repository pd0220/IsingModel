// 2D Ising model simulation via Metropolis-Hastings algorithm
// parallel setup ~ single checkboard: preventing race conditions

// include header(s)

#include <hip/hip_runtime.h>
#include <random>
#include <cmath>
#include <numeric>
#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <stdio.h>
// time measurement
#include <chrono>
// cuRAND
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// constants
// spatial size of simulation table (use > 1 and even)
const int spatialSize = 1024;
// integration time
const int intTime = (int)1e4;
// coupling
const float coupling = (float)0.45;
// file name to save data
const std::string fileName = "C:\\Users\\david\\Desktop\\MSc\\Ising model\\RENORM_HW1\\magnetisation.txt";
// number of threads per block
const int nThread = 64;
// block size
const int sizeInBlocks = 16;
// number of blocks
const int nBlock = sizeInBlocks * sizeInBlocks;
// size of a single block
const int blockSize = spatialSize / sizeInBlocks;

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate the energy difference due to a single flip
__device__ int DeltaE(int *table, int row, int col, int dim)
{
    // spin in question
    int s = table[row * dim + col];

    // periodic boundary conditions
    int rowRight = (row + 1) % dim, rowLeft = (row + dim - 1) % dim, colDown = (col + 1) % dim, colUp = (col + dim - 1) % dim;

    // neighbours
    int right = table[rowRight * spatialSize + col], left = table[rowLeft * spatialSize + col], down = table[row * spatialSize + colDown], up = table[row * spatialSize + colUp];

    // return energy difference (divided by J)
    return 2 * s * (up + down + left + right);
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate rate
__device__ float Rate(int *table, int row, int col, int dim, float coupling)
{
    // energy difference due to flip (divided by J)
    int deltaE = DeltaE(table, row, col, dim);
    // calculate rate
    if (deltaE < 0)
        return 1.;
    else if (deltaE == 0)
        return 0.5;
    else
        return expf(-coupling * deltaE);
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// square function for integers
__host__ __device__ int Square(int x) { return x * x; }

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// spin flip ~ site visit for given (row, col)
__device__ void SpinFlip(int *table, float coupling, hiprandState &state, int row, int col)
{
    // random number for flipping
    float randVal = hiprand_uniform(&state);
    // rate
    float rate = Rate(table, row, col, spatialSize, coupling);
    // flip or not to flip...
    if (rate > randVal)
        table[row * spatialSize + col] *= -1;
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// kernel for Metropolis sweep ~ even sites
__global__ void KernelMetropolisEven(int *table, hiprandState *states, float coupling, int sweep)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(2 * sweep, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * blockSize;
    int minCol = bid * blockSize - sizeInBlocks * minRow;
    // move to thread
    minRow += id * blockSize / nThread;

    for (int irow = minRow; irow < minRow + blockSize / nThread; irow++)
    {
        // columns for even sites only
        for (int icol = (((irow % 2) == 0) ? minCol : minCol + 1); icol < minCol + blockSize; icol += 2)
        {
            SpinFlip(table, coupling, states[tid], irow, icol);
        }
    }
}

// kernel for Metropolis sweep ~ odd sites
__global__ void KernelMetropolisOdd(int *table, hiprandState *states, float coupling, int sweep)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(2 * sweep + 1, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * blockSize;
    int minCol = bid * blockSize - sizeInBlocks * minRow;
    // move to thread
    minRow += id * blockSize / nThread;

    for (int irow = minRow; irow < minRow + blockSize / nThread; irow++)
    {
        // columns for odd sites only
        for (int icol = (((irow % 2) == 0) ? minCol + 1 : minCol); icol < minCol + blockSize; icol += 2)
        {
            SpinFlip(table, coupling, states[tid], irow, icol);
        }
    }
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// main function
int main(int, char **)
{
    // random number generation
    std::random_device rd{};
    std::mt19937 gen(rd());
    // [0, 1] ~ real
    std::uniform_real_distribution<double> distrReal(0., 1.);

    // vector of time measurements
    std::vector<float> timeMeasurement;

    // initialize spins (cold start)
    // host
    std::vector<int> table(Square(spatialSize), 1);
    // device
    int *tableDev = nullptr;
    // cuRAND states
    hiprandState *statesDev = nullptr;

    // container for magnetisation values
    std::vector<double> m(intTime, 0.);

    // simulation
    // Metropolis sweeps
    for (int iSweep = 0; iSweep < intTime; iSweep++)
    {
        // device
        tableDev = nullptr;
        // cuRAND states
        statesDev = nullptr;

        // CUDA error handling
        hipError_t err = hipSuccess;

        // memory allocation for the device
        err = hipMalloc((void **)&tableDev, Square(spatialSize) * sizeof(int));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipMalloc((void **)&statesDev, nBlock * nThread * sizeof(hiprandState));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // copy data onto device
        err = hipMemcpy(tableDev, table.data(), Square(spatialSize) * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to device (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // even kernel
        KernelMetropolisEven<<<nBlock, nThread>>>(tableDev, statesDev, coupling, iSweep);

        // odd kernel
        KernelMetropolisOdd<<<nBlock, nThread>>>(tableDev, statesDev, coupling, iSweep);

        // get errors from run
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        // copy data from device
        err = hipMemcpy(table.data(), tableDev, Square(spatialSize) * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to host: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // free memory
        err = hipFree(tableDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipFree(statesDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // compute magnetisation
        m[iSweep] = std::accumulate(table.begin(), table.end(), 0.) / Square(spatialSize);
    }

    // write magnetisation results to file
    // file
    std::ofstream file;
    file.open(fileName);
    for (int im = 0; im < intTime; im++)
    {
        file << m[im] << std::endl;
    }
    file.close();
}