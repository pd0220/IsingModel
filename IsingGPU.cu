// 2D Ising model simulation via Metropolis-Hastings algorithm
// parallel setup ~ single checkboard: preventing race conditions

// include header(s)

#include <hip/hip_runtime.h>
#include <random>
#include <cmath>
#include <numeric>
#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <stdio.h>
// time measurement
#include <chrono>
// cuRAND
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// constants
// spatial size of simulation table (use > 1 and even)
const int spatialSize = 256;
// integration time
const int intTime = (int)1e4;
// scale for coupling index
const float scalar = 50.;
// number of threads per block
const int nThread = 64;
// block size
const int sizeInBlocks = 4;
// number of blocks
const int nBlock = sizeInBlocks * sizeInBlocks;
// size of a single block
const int blockSize = spatialSize / sizeInBlocks;

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate the energy difference due to a single flip
__device__ int DeltaE(int *table, int row, int col, int dim)
{
    // spin in question
    int s = table[row * dim + col];

    // periodic boundary conditions
    int rowRight = (row + 1) % dim, rowLeft = (row + dim - 1) % dim, colDown = (col + 1) % dim, colUp = (col + dim - 1) % dim;

    // neighbours
    int right = table[rowRight * spatialSize + col], left = table[rowLeft * spatialSize + col], down = table[row * spatialSize + colDown], up = table[row * spatialSize + colUp];

    // return energy difference (divided by J)
    return 2 * s * (up + down + left + right);
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate rate
__device__ float Rate(int *table, int row, int col, int dim, float coupling)
{
    // energy difference due to flip (divided by J)
    int deltaE = DeltaE(table, row, col, dim);
    // calculate rate
    if (deltaE < 0)
        return 1.;
    else if (deltaE == 0)
        return 0.5;
    else
        return expf(-coupling * deltaE);
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// square function for integers
__host__ __device__ int Square(int x) { return x * x; }

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// spin flip ~ site visit for given (row, col)
__device__ void SpinFlip(int *table, float coupling, hiprandState &state, int row, int col)
{
    // random number for flipping
    float randVal = hiprand_uniform(&state);
    // rate
    float rate = Rate(table, row, col, spatialSize, coupling);
    // flip or not to flip...
    if (rate > randVal)
        table[row * spatialSize + col] *= -1;
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// kernel for Metropolis sweep ~ even sites
__global__ void KernelMetropolisEven(int *table, hiprandState *states, float coupling, int sweep)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(2 * sweep, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * blockSize;
    int minCol = bid * blockSize - sizeInBlocks * minRow;
    // move to thread
    minRow += id * blockSize / nThread;

    for (int irow = minRow; irow < minRow + blockSize / nThread; irow++)
    {
        // columns for even sites only
        for (int icol = (((irow % 2) == 0) ? minCol : minCol + 1); icol < minCol + blockSize; icol += 2)
        {
            SpinFlip(table, coupling, states[tid], irow, icol);
        }
    }
}

// kernel for Metropolis sweep ~ odd sites
__global__ void KernelMetropolisOdd(int *table, hiprandState *states, float coupling, int sweep)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(2 * sweep + 1, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * blockSize;
    int minCol = bid * blockSize - sizeInBlocks * minRow;
    // move to thread
    minRow += id * blockSize / nThread;

    for (int irow = minRow; irow < minRow + blockSize / nThread; irow++)
    {
        // columns for odd sites only
        for (int icol = (((irow % 2) == 0) ? minCol + 1 : minCol); icol < minCol + blockSize; icol += 2)
        {
            SpinFlip(table, coupling, states[tid], irow, icol);
        }
    }
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// main function
int main(int, char **)
{
    // random number generation
    std::random_device rd{};
    std::mt19937 gen(rd());
    // [0, 1] ~ real
    std::uniform_real_distribution<double> distrReal(0., 1.);
    // random generator lambda for spin initialisation
    auto RandSpin = [&distrReal, &gen]()
    { return (float)distrReal(gen) > 0.5 ? 1 : -1; };

    // vector of time measurements
    std::vector<float> timeMeasurement;

    // initialize spins
    // host
    std::vector<int> table(Square(spatialSize));
    // device
    int *tableDev = nullptr;
    // cuRAND states
    hiprandState *statesDev = nullptr;

    // file
    std::ofstream file;
    file.open((std::string) "C:\\Users\\david\\Desktop\\MSc\\Ising model\\Python\\testGPU.txt");

    // loop over couplings
    for (int iCoupling = 0; iCoupling < 100; iCoupling += 5)
    {
        // real coupling
        float coupling = (float)(iCoupling / scalar);

        // (re)initialize spins
        // host
        //std::generate(table.begin(), table.end(), RandSpin);
        table = std::vector<int>(Square(spatialSize), 1);

        // CUDA time measurement
        hipEvent_t evt[2];
        hipError_t err = hipSuccess;
        for (auto &e : evt)
        {
            err = hipEventCreate(&e);
            if (err != hipSuccess)
            {
                std::cout << "Error in creating time measurement object: " << hipGetErrorString(err) << std::endl;
                return -1;
            }
        }

        // device
        tableDev = nullptr;
        // cuRAND states
        statesDev = nullptr;

        // memory allocation for the device
        err = hipMalloc((void **)&tableDev, Square(spatialSize) * sizeof(int));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipMalloc((void **)&statesDev, nBlock * nThread * sizeof(hiprandState));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // copy data onto device
        err = hipMemcpy(tableDev, table.data(), Square(spatialSize) * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to device (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // TIME #0
        err = hipEventRecord(evt[0]);
        if (err != hipSuccess)
        {
            std::cout << "Error recording time at start: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // simulation
        // Metropolis sweeps
        for (int iSweep = 0; iSweep < intTime; iSweep++)
        {
            // even kernel
            KernelMetropolisEven<<<nBlock, nThread>>>(tableDev, statesDev, coupling, iSweep);

            // odd kernel
            KernelMetropolisOdd<<<nBlock, nThread>>>(tableDev, statesDev, coupling, iSweep);

            // get errors from run
            err = hipGetLastError();
            if (err != hipSuccess)
            {
                std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << std::endl;
                return -1;
            }
        }

        // TIME #1
        err = hipEventRecord(evt[1]);
        if (err != hipSuccess)
        {
            std::cout << "Error recording time at stop: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // copy data from device
        err = hipMemcpy(table.data(), tableDev, Square(spatialSize) * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to host: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // free memory
        err = hipFree(tableDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipFree(statesDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // CUDA time measurement
        err = hipEventSynchronize(evt[1]);
        if (err != hipSuccess)
        {
            std::cout << "Error in synchronising time measurements: " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        // time in milliseconds
        float dt = 0.;
        err = hipEventElapsedTime(&dt, evt[0], evt[1]);
        if (err != hipSuccess)
        {
            std::cout << "Error in calculating total elapsed time: " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        for (auto &e : evt)
        {
            err = hipEventDestroy(e);
            if (err != hipSuccess)
            {
                std::cout << "Error in deleting time measurement object: " << hipGetErrorString(err) << std::endl;
                return -1;
            }
        }
        timeMeasurement.push_back(dt);

        // averaging magnetisation
        file << coupling << " " << std::accumulate(table.begin(), table.end(), 0.) / Square(spatialSize) << std::endl;
    }
    file.close();

    // print computation time
    std::cout << "Mean parrallel computation time for a single table on GPU: "
              << std::accumulate(timeMeasurement.begin(), timeMeasurement.end(), 0.) / static_cast<double>(timeMeasurement.size()) << " ms." << std::endl;
}