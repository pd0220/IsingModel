// 2D Ising model simulation via Metropolis-Hastings algorithm
// parallel setup ~ single checkboard: preventing race conditions

// include header(s)

#include <hip/hip_runtime.h>
#include <random>
#include <cmath>
#include <numeric>
#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
// time measurement
#include <chrono>
// cuRAND
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// constants
// spatial size of simulation table (use > 1 and even)
const int spatialSize = 128;
// integration time
const int intTime = (int)2e6;
// scale for coupling index
const float scalar = 50.;
// number of threads per block
const int nThread = 16;
// block size
const int sizeInBlocks = 4;
// number of blocks
const int nBlock = sizeInBlocks * sizeInBlocks;

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate the sign of the energy difference due to a single flip
__device__ int DeltaE(int *table, int row, int col, int dim)
{
    // spin in question
    int s = table[row * spatialSize + col];

    // periodic boundary conditions
    int rowRight = (row + 1) % dim, rowLeft = (row + dim - 1) % dim, colDown = (col + 1) % dim, colUp = (col + dim - 1) % dim;

    // neighbours
    int right = table[rowRight * spatialSize + col], left = table[rowLeft * spatialSize + col], down = table[row * spatialSize + colDown], up = table[row * spatialSize + colUp];

    // quantity proportional to energy difference
    int energy = s * (up + down + left + right);

    // return sign of difference or zero (initialize to zero)
    int sign = 0;
    if (energy > 0)
        sign = 1;
    else if (energy < 0)
        sign = -1;
    return sign;
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// calculate rate
__device__ float Rate(int *table, int row, int col, int dim, float coupling)
{
    // sign of energy difference due to flip
    int deltaE = DeltaE(table, row, col, dim);
    // calculate rate
    if (deltaE < 0)
        return 1.;
    else if (deltaE == 0)
        return 0.5;
    else
        return std::exp(-2 * coupling * deltaE);
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// square function for integers
__host__ __device__ int Square(int x) { return x * x; }

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// spin flip ~ site visit for given (row, col)
__device__ void SpinFlip(int *table, bool parity, float coupling, hiprandState &state, int row, int col)
{
    // random number for flipping
    float randVal = hiprand_uniform(&state);
    // rate
    float rate = Rate(table, row, col, spatialSize, coupling);
    // flip or not to flip
    if (rate > randVal)
        table[row * spatialSize + col] *= -1;
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// kernel for Metropolis sweep ~ even sites
__global__ void KernelMetropolisEven(int *table, hiprandState *states, float coupling)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(42, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * spatialSize / sizeInBlocks;
    int minCol = bid * spatialSize / sizeInBlocks - sizeInBlocks * minRow;
    // move to thread
    minRow += id * spatialSize / sizeInBlocks / nThread;

    __syncthreads();

    for (int irow = minRow; irow < minRow + spatialSize / sizeInBlocks / nThread; irow++)
    {
        for (int icol = (((irow % 2) == 0) ? minCol : minCol + 1); icol < minCol + spatialSize / sizeInBlocks; icol += 2)
        {
            SpinFlip(table, true, coupling, states[tid], irow, icol);
        }
    }
}

// kernel for Metropolis sweep ~ odd sites
__global__ void KernelMetropolisOdd(int *table, hiprandState *states, float coupling)
{
    // thread index inside the block
    int id = threadIdx.x;
    // block index
    int bid = blockIdx.x;
    // thread index
    int tid = bid * blockDim.x + id;
    // initialize cuRAND
    hiprand_init(42, tid, 0, &states[tid]);

    // locate block and thread
    int minRow = (int)(bid / sizeInBlocks) * spatialSize / sizeInBlocks;
    int minCol = bid * spatialSize / sizeInBlocks - sizeInBlocks * minRow;
    // move to thread
    minRow += id * spatialSize / sizeInBlocks / nThread;

    __syncthreads();

    for (int irow = minRow; irow < minRow + spatialSize / sizeInBlocks / nThread; irow++)
    {
        for (int icol = (((irow % 2) == 0) ? minCol + 1 : minCol); icol < minCol + spatialSize / sizeInBlocks; icol += 2)
        {
            SpinFlip(table, true, coupling, states[tid], irow, icol);
        }
    }
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------

// main function
int main(int, char **)
{
    // random number generation
    std::random_device rd{};
    std::mt19937 gen(rd());
    // [0, 1] ~ real
    std::uniform_real_distribution<double> distrReal(0., 1.);
    // random generator lambda for spin initialisation
    auto RandSpin = [&distrReal, &gen]()
    { return (float)distrReal(gen) > 0.5 ? 1 : -1; };

    // initialize spins
    // host
    std::vector<int> table(Square(spatialSize));
    std::generate(table.begin(), table.end(), RandSpin);
    // device
    int *tableDev = nullptr;
    // cuRAND states
    hiprandState *statesDev = nullptr;

    for (int iCoupling = 70; iCoupling < 100; iCoupling += 10)
    {
        // real coupling
        //float coupling = (float)(iCoupling / scalar);
        float coupling = 10.;
        
        // (re)initialize spins
        // host
        std::generate(table.begin(), table.end(), RandSpin);
        // device
        tableDev = nullptr;
        // cuRAND states
        statesDev = nullptr;

        // memory allocation for the device
        hipError_t err = hipSuccess;
        err = hipMalloc((void **)&tableDev, Square(spatialSize) * sizeof(int));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipMalloc((void **)&statesDev, nBlock * nThread * sizeof(hiprandState));
        if (err != hipSuccess)
        {
            std::cout << "Error allocating CUDA memory (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // copy data onto device
        err = hipMemcpy(tableDev, table.data(), Square(spatialSize) * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to device (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // simulation
        // Metropolis sweeps
        for (int iSweep = 0; iSweep < intTime; iSweep++)
        {
            // even kernel
            KernelMetropolisEven<<<nBlock, nThread>>>(tableDev, statesDev, coupling);

            // odd kernel
            KernelMetropolisOdd<<<nBlock, nThread>>>(tableDev, statesDev, coupling);
        }

        // get errors from run
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // copy data from device
        err = hipMemcpy(table.data(), tableDev, Square(spatialSize) * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            std::cout << "Error copying memory to host: " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // free memory
        err = hipFree(tableDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (TABLE): " << hipGetErrorString(err) << std::endl;
            return -1;
        }
        err = hipFree(statesDev);
        if (err != hipSuccess)
        {
            std::cout << "Error freeing allocation (cuRAND): " << hipGetErrorString(err) << std::endl;
            return -1;
        }

        // print coupling
        std::cout << "J = " << coupling;
        // print magnetisation
        std::cout << " |M| = " << std::accumulate(table.begin(), table.end(), 0.) / Square(spatialSize) << std::endl;

        // file
        std::ofstream file;
        file.open((std::string) "C:\\Users\\david\\Desktop\\MSc\\Ising model\\Python\\testGPUTable.txt");
        for (int i = 0; i < spatialSize; i++)
        {
            for (int j = 0; j < spatialSize; j++)
            {
                file << table[i * spatialSize + j] << " ";
            }
            file << std::endl;
        }
        file.close();
    }
}